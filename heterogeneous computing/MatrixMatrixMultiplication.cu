#include "hip/hip_runtime.h"

/*************************************
 * Matrix-Matrix product with CUBLAS *
 *************************************/

#include <stdio.h>
#include <mkl_blas.h>
#include "hipblas.h" /* Write here the name of the CUBLAS header file */

#define CUDA_SAFE_CALL( call ) { cuAssert((call), __FILE__, __LINE__); }
inline void cuAssert( hipError_t err, const char *file, int line, bool abort=true)
{
		if( hipSuccess != err )
		{
				fprintf(stderr, "CUDA: error ocurred in %s %s %d\n", hipGetErrorString(err), file, line );
				if( abort )
						exit( err );

		}
}

#define CUBLAS_SAFE_CALL( call ) { cublasAssert((call), __FILE__, __LINE__); }
inline void cublasAssert( hipblasStatus_t err, const char *file, int line, bool abort=true)
{
		if( HIPBLAS_STATUS_SUCCESS != err )
		{
				fprintf(stderr, "CUBLAS: error ocurred in %s %s %d\n", err, file, line );
				if( abort )
						exit( err );

		}
}

/* Matrices stored by columns: BLAS style */
#define	A(i,j)			A[ (i) + ((j)*(n)) ]
#define	B(i,j)			B[ (i) + ((j)*(n)) ]
#define	C(i,j)			C[ (i) + ((j)*(n)) ]
#define	h_C(i,j)		h_C[ (i) + ((j)*(n)) ]
#define	h_C2(i,j)		h_C2[ (i) + ((j)*(n)) ]
#define	het_C(i,j)	het_C[ (i) + ((j)*(n)) ]
#define	d_A(i,j) 		d_A[ (j) + ((i)*(n)) ]

int main( int argc, char *argv[] ) 
{
		int n, m, nm;
		float weigth;
		unsigned int i, j;

		if( argc < 3 ) 
		{
				printf( "Usage: %s n weight\n", argv[0] );
				exit( -EXIT_FAILURE );
		}

		sscanf( argv[1],"%d",&n );
		sscanf( argv[2],"%f",&weigth );

		m = n * weigth;
		nm = n - m;

		// General matrices
		double *A = (double *) malloc( n * n * sizeof(double) );
		double *B = (double *) malloc( n * n * sizeof(double) );

		// Result matrices
		double *C = (double *) malloc( n * n * sizeof(double) ); 			// CPU execution
		double *h_C = (double *) malloc( n * n * sizeof(double) );	 	// GPU execution
		double *het_C = (double *) malloc( n * n * sizeof(double) );	// Heterogeneous execution

		// GPU matrices
		double *d_A, *d_B, *d_C;

		// Heterogenous matrices
		double *h_C2 = (double *) malloc( n * m * sizeof(double) );

		printf( "%s: Generating two random matrices of size %dx%d...\n", argv[0], n, n );

		for( i = 0; i < n; i++ )
		{
				for( j = 0; j < n; j++ )
						A( i, j ) = 2.0 * ( (double) rand() / RAND_MAX ) - 1.0;

		}

		for( i = 0; i < n; i++ )
		{
				for( j = 0; j < n; j++ )
						B( i, j ) = 2.0 * ( (double) rand() / RAND_MAX ) - 1.0;

		}

		/* STARTUP CUBLAS context */
		hipblasHandle_t handle;
		CUBLAS_SAFE_CALL( hipblasCreate( &handle ) );

		hipEvent_t start, stop;
		CUDA_SAFE_CALL( hipEventCreate( &start ) );
		CUDA_SAFE_CALL( hipEventCreate( &stop ) );

		const char trans = 'N';
		const double ONE = 1.0;
		const double ZERO = 0.0;

		// MKL execution (CPU)
		printf( "%s: C = A * B in CPU...\n", argv[0] );
		CUDA_SAFE_CALL( hipEventRecord(start, NULL) );
		dgemm( &trans, &trans, &n, &n, &n, &ONE, A, &n, B, &n, &ZERO, C, &n );
		CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );
		CUDA_SAFE_CALL( hipEventSynchronize( stop ) );
		float msecCPU = 0.0f;
		CUDA_SAFE_CALL( hipEventElapsedTime( &msecCPU, start, stop ) );

		// CuBLAS execution (GPU)
		printf( "%s: C = A * B in GPU...\n", argv[0] );
		CUDA_SAFE_CALL( hipMalloc((void **) &d_A, n * n * sizeof(double) ) );
		CUDA_SAFE_CALL( hipMalloc((void **) &d_B, n * n * sizeof(double) ) );
		CUDA_SAFE_CALL( hipMalloc((void **) &d_C, n * n * sizeof(double) ) );
		CUBLAS_SAFE_CALL( hipblasSetMatrix( n, n, sizeof(double), A, n, d_A, n ) );
		CUBLAS_SAFE_CALL( hipblasSetMatrix( n, n, sizeof(double), B, n, d_B, n ) );

		CUDA_SAFE_CALL( hipEventRecord(start, NULL) );
		CUBLAS_SAFE_CALL( hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &ONE, d_A, n, d_B, n, &ZERO, d_C, n ) );
		CUDA_SAFE_CALL( hipEventRecord( stop, NULL ) );
		CUDA_SAFE_CALL( hipEventSynchronize( stop ) );
		CUBLAS_SAFE_CALL( hipblasGetMatrix( n, n, sizeof(double), d_C, n, h_C, n ) );
		float msecGPU = 0.0f;
		CUDA_SAFE_CALL( hipEventElapsedTime( &msecGPU, start, stop ) );
		CUDA_SAFE_CALL( hipFree( d_B ) );
		CUDA_SAFE_CALL( hipFree( d_C ) );

		// Heterogeneous execution (CPU + GPU)
		printf( "%s: C = A * B in CPU + GPU...\n",argv[0] );
		CUDA_SAFE_CALL( hipMalloc((void **) &d_B, n * m * sizeof(double) ) );
		CUDA_SAFE_CALL( hipMalloc((void **) &d_C, n * m * sizeof(double) ) );
		CUBLAS_SAFE_CALL( hipblasSetMatrix( n, m, sizeof(double), &B(0,nm), n, d_B, n ) );
		CUDA_SAFE_CALL( hipEventRecord( start, NULL ) );
		CUBLAS_SAFE_CALL( hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, n, &ONE, d_A, n, d_B, n, &ZERO, d_C, n ) );
		dgemm( &trans, &trans, &n, &nm, &n, &ONE, A, &n, B, &n, &ZERO, het_C, &n );
		CUBLAS_SAFE_CALL( hipblasGetMatrix( n, m, sizeof(double), d_C, n, h_C2, n ) );
		memcpy( &het_C(0, nm), h_C2, n * m * sizeof(double) );
		CUDA_SAFE_CALL( hipEventRecord( stop, NULL ) );
		CUDA_SAFE_CALL( hipEventSynchronize( stop ) );
		float msecCPUGPU = 0.0f;
		CUDA_SAFE_CALL( hipEventElapsedTime( &msecCPUGPU, start, stop ) );

		int one = 1;
		int maxid = idamax( &n, C, &one );
		double max = C[maxid];
		double error = ZERO;

		for( j = 1; j < n; j++ )
		{
				for( i = 1; i < n; i++ )
				{
						double a = fabs( C( i, j ) - h_C( i, j ) ) / max;
						error = a > error ? a : error;
				}
		}

		printf( "Error CPU/GPU = %.3e\n",error );

		one = 1;
		maxid = idamax( &n, C, &one );
		max = C[maxid];
		error = ZERO;

		for( j = 1; j < n; j++ )
		{
				for( i = 1; i < n; i++ )
				{
						double a = fabs( C( i, j ) - het_C( i, j ) ) / max;
						error = a > error ? a : error;
				}
		}

		printf( "Error CPU/CPU + GPU = %.3e\n",error );

		double flops = 2.0 * (double) n * (double) n * (double) n;

		float gigaFlopsCPU = ( flops * 1.0e-9f ) / ( msecCPU / 1000.0f );
		float gigaFlopsGPU = ( flops * 1.0e-9f ) / ( msecGPU / 1000.0f );
		float gigaFlopsCPUGPU = ( flops * 1.0e-9f ) / ( msecCPUGPU / 1000.0f );

		printf( "CPU time = %.2f msec.\n", msecCPU );
		printf( "GPU time = %.2f msec.\n", msecGPU );
		printf( "CPU + GPU time = %.2f msec.\n", msecCPUGPU );
		printf( "GFlops CPU = %.2f \n", gigaFlopsCPU );
		printf( "GFlops GPU = %.2f \n", gigaFlopsGPU );
		printf( "GFlops CPU + GPU = %.2f \n", gigaFlopsCPUGPU );

		// CPU matrices
		free( A );
		free( B );
		free( C );
		//free( h_C );
		free( het_C );
		free( h_C2 );

		//GPU matrices
		hipFree( d_A );
		hipFree( d_B );
		hipFree( d_C );
		hipblasDestroy( handle );
}

